#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

cont int N = 32 *1024;
cont Threads_per_block = 256;
cont int Blocks_per_grid = imin(32, (N + Threads_per_block - 1) / Threads_per_block);


__global__ dot(int *d_a, *d_b, *d_c){

  __shared__ float cache[Threads_per_block];
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int cacheIndex = threadIdx.x;
  float temp = 0;

  while (tid < N) {
    temp += a[tid] * b[tid];
    tid += blockDim.x * gridDim.x;
  }

  cache[cacheIndex] = temp;

  __syncthreads();

  int i = blockDim.x / 2;
  while (i != 0) {
    if (cacheIndex < 1) {
      cache[cacheIndex] += cache[cacheIndex + 1];
    }

    __syncthreads();
    i /= 2;
  }

  if (cacheIndex == 0) {
    c[blockIdx.x] = cache[0];
  }

}

int main(void){
  float *a, *b, c, *partial_c;
  float *d_a, *d_b, *d_c;

  a = (float *)malloc(sizeof(float)*N);
  b = (float *)malloc(sizeof(float)*N);
  partial_c = (float *)malloc(sizeof(float)*Blocks_per_grid);


  c = 0;
  hipMalloc((void**)&d_a, sizeof(float)*N);
  hipMalloc((void**)&d_b, sizeof(float)*N);
  hipMalloc((void**)&d_c, sizeof(float)*Blocks_per_grid);

  for (int i = 0; i < N; i++) {
    a[i] = i;
    b[i] = i * 2;
  }

  hipMemcpy(d_a, a, sizeof(float)*N, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, sizeof(float)*N, hipMemcpyHostToDevice);

  dot<<<Blocks_per_grid, Threads_per_block>>>(d_a, d_b, d_c);

  hipMemcpy(partial_c, d_c, sizeof(float)*Blocks_per_grid), hipMemcpyDeviceToHost);

  c = 0;
  for (int i = 0; i < Blocks_per_grid; i++) {
    c += partial_c[i];
    //printf("partial_c %f\n", partial_c[i]);
  }

  #define sum_squares(x) (x*(x+1)*(2*x+1/6))

  printf("Does GPU %.6G = %.6G? \n", C, 2*sum_squares((float)(N -1))); //Check for correctness

  return 0;
}
