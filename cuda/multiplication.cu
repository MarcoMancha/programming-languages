/*
	Marco Mancha A01206194
	Lab 2 CUDA
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void valid_multiplication(int *res, int *mat2, int r2, int c2, int *mat1, int r1, int c1){
	// Obtain row and col of the grid
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;

	//Obtain position of actual thread
	int id = row * r1 + col;
	int sum = 0;

	//Check if we are inside the grid
	if (row < r1 && col < c2) {
		//Make the sum of the multiplications of the rows * columns
		for(int i = 0; i < r1; i++) {
			sum = sum + mat1[row * r1 + i] * mat2[i * r1 + col];
		}
		//Assign value to new matrix
	 	res[id] = sum;
	}
}


void print_mat(int *mat, int n, int m){
	for (int i = 0; i < n; i++){
		for (int j = 0; j < m; j++){
			printf("%d\t", mat[i*n+j]);
		}
		printf("\n");
	}
	printf("\n");
}

void fill_mat(int *mat, int n, int m){
	int acum;
	for (int i = 0; i < n; i++){
		acum = 0;
		for (int j = 0; j < m; j++){
			acum++;
			mat[i * n + j] = acum;
		}
	}
}

int main(){
	int *h_matrix3, *h_matrix1, *h_matrix2;
	int *d_matrix3, *d_matrix1, *d_matrix2;
	int r1, c1 = 0, r2 = 1, c2;

	// Read matrix size (I couldn't use scanf)
	int flag;
  while(c1 != r2){
    printf("Size matrix 1 \n");
    scanf("rows: %d%*c", &r1);
    scanf("columns: %d%*c", &c1);

    printf("Size matrix 2 \n");
    scanf("rows: %d%*c", &r2);
    scanf("columns: %d%*c", &c2);
    flag++;
  }

	// Space on host for matrices
	h_matrix3 = (int *)malloc(sizeof(int)*r1*c2);
	h_matrix1 = (int *)malloc(sizeof(int)*r1*c1);
	h_matrix2 = (int *)malloc(sizeof(int)*r2*c2);

	// Fill first 2 matrix
	fill_mat(h_matrix1, r1, c1);
	fill_mat(h_matrix2, r2, c2);

  print_mat(h_matrix1, r1, c1);
	print_mat(h_matrix2, r2, c2);

	// Space on device for 3 matrices
	hipMalloc((void**)&d_matrix3,sizeof(int)*r1*c2);
	hipMalloc((void**)&d_matrix1,sizeof(int)*r1*c1);
	hipMalloc((void**)&d_matrix2,sizeof(int)*r2*c2);

	// Copy values from host to device matrix
	hipMemcpy(d_matrix1, h_matrix1,sizeof(int)*r1*c1, hipMemcpyHostToDevice);
	hipMemcpy(d_matrix2, h_matrix2,sizeof(int)*r2*c2, hipMemcpyHostToDevice);

	int ThreadsPerBlock = r1;
	int NumBlocks = (ThreadsPerBlock + (r1*c2))/ThreadsPerBlock;

	// Create the grid
	dim3 Blocks(NumBlocks,NumBlocks);
	dim3 Threads(ThreadsPerBlock,ThreadsPerBlock);

	// Call cuda function
	valid_multiplication<<<Blocks, Threads>>>(d_matrix3, d_matrix2, r2, c2, d_matrix1, r1, c1);

	// Copy results
	hipMemcpy(h_matrix3, d_matrix3,sizeof(int)*r1*c2, hipMemcpyDeviceToHost);

	// Print and free matrices
	print_mat(h_matrix3, r1, c2);

	free(h_matrix3);
	free(h_matrix2);
	free(h_matrix1);

	hipFree(d_matrix3);
	hipFree(d_matrix2);
	hipFree(d_matrix1);

}
