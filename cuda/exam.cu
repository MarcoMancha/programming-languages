// Marco Mancha A01206194
// template provided for cuda quizz 3.
// remember to write your own comments in the code below.


#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 9		//size of original matrix
#define K N/3		//size of compressed matrrix

__global__ void compress(float *mat, int n, float *comp, int k){

  // row and col from grid
  int row = threadIdx.y + blockIdx.y * blockDim.y;
  int col = threadIdx.x + blockIdx.x * blockDim.x;

  if(row < k && col < k){
    //initialize value of comp
    comp[col * k + row] = 0;
    for(int i = 0 ; i < k ; i++){
      for(int j = 0 ; j < k ; j++){
        // Sum of the elements
        comp[col + row * k] += mat[(col + j) + (row + i) * n];
        printf("val %i: %f\n", col * k + row, mat[(col + j) + (row + i) * n]);
      }
    }
    printf("ELEMENTO %i: %f\n", col * k + row, comp[col * k + row]);
    // Average of each section
    comp[col * k + row] = comp[col * k + row] / n;
  }
}

void print_mat(float *mat, int n){
	for (int i = 0; i < n; i++){
		for (int j = 0; j < n; j++){
			printf("%.1f\t", mat[i*n+j]);
		}
		printf("\n");
	}
	printf("\n");
}


void fill_mat(float *mat, int n){
	int c = 0;
	for (int i = 0; i < n; i++){
		for (int j = 0; j < n; j++){
			mat[i*n+j] = c++;
		}
	}
}

int main(){
	float *h_compress, *h_matrix;
	float *d_compress, *d_matrix;

  //Reserve space on host
	h_compress = (float *)malloc(sizeof(float)*K*K);
	h_matrix = (float *)malloc(sizeof(float)*N*N);

	fill_mat(h_matrix, N);

  printf("\n input mat \n");
	print_mat(h_matrix, N);

  // Reserve space in cuda
  hipMalloc((void**)&d_matrix,sizeof(float)*N*N);
	hipMalloc((void**)&d_compress,sizeof(float)*K*K);

  // Copy from host to device
	hipMemcpy(d_matrix, h_matrix,sizeof(float)*N*N, hipMemcpyHostToDevice);
	hipMemcpy(d_compress, h_compress,sizeof(float)*K*K, hipMemcpyHostToDevice);

  int ThreadsPerBlock = 1;	 // choose wisely
  int NumBlocks = K;

  // Define grid
	dim3 Blocks(NumBlocks,NumBlocks);
	dim3 Threads(ThreadsPerBlock,ThreadsPerBlock);

  // Call cuda function
  compress<<<Blocks, Threads>>>(d_matrix, N, d_compress, K);

  // COpy from device to host
  hipMemcpy(h_compress, d_compress,sizeof(float)*K*K, hipMemcpyDeviceToHost);

  printf("\n compress mat \n");
	print_mat(h_compress, K);

  // Free elements from host and device
	free(h_matrix);
	free(h_compress);

	hipFree(d_matrix);
	hipFree(d_compress);

}
